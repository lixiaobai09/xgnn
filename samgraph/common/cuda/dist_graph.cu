#include "dist_graph.h"

#include <sys/mman.h>
#include <sys/unistd.h>
#include <sys/wait.h>

#include <cstring>
#include <iomanip>
#include <set>
#include <algorithm>

#include "../device.h"
#include "../timer.h"


namespace samgraph {
namespace common {
namespace cuda {

namespace {

template<typename T>
std::set<T> operator- (const std::set<T> &a, const std::set<T> &b) {
  std::set<T> ret;
  for (auto i : a) {
    if (!b.count(i)) {
      ret.insert(i);
    }
  }
  return std::move(ret);
};

}; // namespace

std::shared_ptr<DistGraph> DistGraph::_inst = nullptr;

void DistGraph::_DatasetPartition(const Dataset *dataset, Context ctx,
    IdType part_id, IdType num_part) {
  auto indptr_data = dataset->indptr->CPtr<IdType>();
  auto indices_data = dataset->indices->CPtr<IdType>();
  IdType num_node = dataset->num_node;
  IdType part_edge_count = 0;
  for (IdType i = part_id; i < num_node; i += num_part) {
    IdType num_edge = (indptr_data[i + 1] - indptr_data[i]);
    part_edge_count += num_edge;
  }

  IdType indptr_size = (num_node / num_part +
      (part_id < num_node % num_part? 1 : 0) + 1);
  _part_indptr[part_id] = Tensor::Empty(kI32, {indptr_size}, CPU(),
      "indptr in device:" + std::to_string(ctx.device_id));
  _part_indices[part_id] = Tensor::Empty(kI32, {part_edge_count}, CPU(),
      "indices in device:" + std::to_string(ctx.device_id));
  part_edge_count = 0;

  for (IdType i = part_id; i < num_node; i += num_part) {
    IdType num_edge = (indptr_data[i + 1] - indptr_data[i]);
    IdType real_id = (i / num_part);
    _part_indptr[part_id]->Ptr<IdType>()[real_id] = part_edge_count;
    std::memcpy(
        &_part_indices[part_id]->Ptr<IdType>()[part_edge_count],
        &indices_data[indptr_data[i]],
        num_edge * sizeof(IdType));
    part_edge_count += num_edge;
  }
  _part_indptr[part_id]->Ptr<IdType>()[indptr_size - 1] = part_edge_count;

  _part_indptr[part_id] = Tensor::CopyTo(_part_indptr[part_id], ctx,
      nullptr, Constant::kAllocNoScale);
  _part_indices[part_id] = Tensor::CopyTo(_part_indices[part_id], ctx,
      nullptr, Constant::kAllocNoScale);
}

void DistGraph::DatasetLoad(Dataset *dataset, int sampler_id,
    Context sampler_ctx) {

  CHECK(sampler_ctx == _group_configs[sampler_id].ctx);
  _sampler_id = sampler_id;

  auto part_ids = _group_configs[sampler_id].part_ids;
  auto ctx_group = _group_configs[sampler_id].ctx_group;
  IdType num_part = ctx_group.size();
  _part_indptr.clear();
  _part_indptr.resize(num_part, nullptr);
  _part_indices.clear();
  _part_indices.resize(num_part, nullptr);

  for (IdType part_id : part_ids) {
    _DatasetPartition(dataset, sampler_ctx, part_id, num_part);
  }

  auto DataIpcShare = [&](std::vector<TensorPtr> &part_data,
      std::vector<size_t> part_size_vec,
      std::string name) {

    {
      for (IdType part_id : part_ids) {
        // share self data to others
        CHECK(sampler_ctx == part_data[part_id]->Ctx());
        CHECK(part_size_vec[part_id] == part_data[part_id]->Shape()[0]);
        auto shared_data = part_data[part_id]->CPtr<IdType>();
        hipIpcMemHandle_t &mem_handle =
          _shared_data->mem_handle[sampler_ctx.device_id][part_id];
        CUDA_CALL(hipIpcGetMemHandle(&mem_handle, (void*)shared_data));
      }
    }
    _Barrier();

    // receive data from others
    for (int i = 0; i < num_part; ++i) {
      if (part_data[i] != nullptr) {
        continue;
      }
      auto ctx = ctx_group[i];
      hipIpcMemHandle_t &mem_handle = _shared_data->mem_handle[ctx.device_id][i];
      void *ptr;
      CUDA_CALL(hipIpcOpenMemHandle(
            &ptr, mem_handle, hipIpcMemLazyEnablePeerAccess));
      part_data[i] = Tensor::FromBlob(ptr, kI32, {part_size_vec[i]}, ctx,
          name + " in device:" + std::to_string(ctx.device_id));
    }
    _Barrier();

  };

  IdType num_node = dataset->num_node;
  std::vector<size_t> part_size_vec(num_part);
  for (size_t i = 0; i < num_part; ++i) {
    part_size_vec[i] = (num_node / num_part +
        (i < num_node % num_part? 1 : 0) + 1);
  }
  DataIpcShare(_part_indptr, part_size_vec, "dataset part indptr");

  part_size_vec.clear();
  part_size_vec.resize(num_part, 0);
  auto indptr_data = dataset->indptr->CPtr<IdType>();
  for (IdType i = 0; i < num_node; ++i) {
    IdType num_edge = indptr_data[i + 1] - indptr_data[i];
    IdType tmp_part_id = (i % num_part);
    part_size_vec[tmp_part_id] += num_edge;
  }
  DataIpcShare(_part_indices, part_size_vec, "dataset part indices");

  CUDA_CALL(hipMalloc((void **)&_d_part_indptr, num_part * sizeof(IdType *)));
  CUDA_CALL(hipMalloc((void **)&_d_part_indices, num_part * sizeof(IdType *)));

  IdType **h_part_indptr, **h_part_indices;
  CUDA_CALL(hipHostMalloc(&h_part_indptr, num_part * sizeof(IdType*)));
  CUDA_CALL(hipHostMalloc(&h_part_indices, num_part * sizeof(IdType*)));
  for (IdType i = 0; i < num_part; i++) {
    h_part_indptr[i] = _part_indptr[i]->Ptr<IdType>();
    h_part_indices[i] = _part_indices[i]->Ptr<IdType>();
  }
  CUDA_CALL(hipMemcpy(_d_part_indptr, h_part_indptr, sizeof(IdType *) * num_part, hipMemcpyDefault));
  CUDA_CALL(hipMemcpy(_d_part_indices, h_part_indices, sizeof(IdType *) * num_part, hipMemcpyDefault));

  CUDA_CALL(hipHostFree(h_part_indptr));
  CUDA_CALL(hipHostFree(h_part_indices));

  _num_node = dataset->num_node;
}

DeviceDistGraph DistGraph::DeviceHandle() const {
  return DeviceDistGraph(
      _d_part_indptr, _d_part_indices,
      _group_configs[_sampler_id].ctx_group.size(),
      _num_node);
}

DistGraph::DistGraph(std::vector<Context> ctxes) {

  PartitionSolver solver(ctxes);
  _group_configs = solver.solve();
  for (auto &config : _group_configs) {
    LOG(INFO) << config;
  }

  int num_worker = ctxes.size();
  _sampler_id = static_cast<int>(Constant::kEmptyKey);

  _shared_data = static_cast<SharedData*>(mmap(NULL, sizeof(SharedData),
                      PROT_READ|PROT_WRITE, MAP_SHARED|MAP_ANONYMOUS, -1, 0));
  CHECK_NE(_shared_data, MAP_FAILED);
  pthread_barrierattr_t attr;
  pthread_barrierattr_init(&attr);
  pthread_barrierattr_setpshared(&attr, PTHREAD_PROCESS_SHARED);
  pthread_barrier_init(&_shared_data->barrier, &attr, num_worker);
}

void DistGraph::_Barrier() {
  int err = pthread_barrier_wait(&_shared_data->barrier);
  CHECK(err == PTHREAD_BARRIER_SERIAL_THREAD || err == 0);
}

void DistGraph::Release(DistGraph *dist_graph) {
  if (dist_graph->_sampler_id != Constant::kEmptyKey) {
    for (int i = 0; i < dist_graph->_part_indptr.size(); i++) {
      if (i != dist_graph->_sampler_id) {
        CUDA_CALL(hipIpcCloseMemHandle(dist_graph->_part_indptr[i]->MutableData()));
        CUDA_CALL(hipIpcCloseMemHandle(dist_graph->_part_indices[i]->MutableData()));
      }
    }
    LOG(INFO) << "Release DistGraph" << " " << dist_graph->_sampler_id;
    // pthread_barrier_wait(&dist_graph->_shared_data->barrier);

    CUDA_CALL(hipFree((void*)dist_graph->_d_part_indptr));
    CUDA_CALL(hipFree((void*)dist_graph->_d_part_indices));
  }
  pthread_barrier_destroy(&dist_graph->_shared_data->barrier);
  munmap(dist_graph->_shared_data, sizeof(SharedData));
}

void DistGraph::Create(std::vector<Context> ctxes) {
  CHECK(_inst == nullptr);
  _inst = std::shared_ptr<DistGraph>(
      new DistGraph(ctxes), Release);
}


PartitionSolver::PartitionSolver(const std::vector<Context> &ctxes)
  : _ctxes(ctxes) {
  std::set<int> set;
  for (auto&ctx : ctxes) {
    set.insert(ctx.device_id);
  }
  CHECK_EQ(set.size(), ctxes.size());
  CHECK_EQ(*set.rbegin() + 1, set.size());
  DetectTopo();
}

void PartitionSolver::DetectTopo() {
  Timer t1;
  // shared memory for transfer detect result
  LinkTopoInfo *shared_data = (LinkTopoInfo*)mmap(NULL, sizeof(LinkTopoInfo), 
    PROT_WRITE | PROT_READ, MAP_SHARED | MAP_ANONYMOUS, -1, 0);
  int pid = fork();
  CHECK(pid != -1);
  if (pid == 0) {
    DetectTopo_child(shared_data);
  } else {
    int wstatus;
    waitpid(pid, &wstatus, 0);
    if (WEXITSTATUS(wstatus) != 0 || WIFSIGNALED(wstatus)) {
      CHECK(false);
    }
    std::memcpy(&_topo_info, shared_data, sizeof(LinkTopoInfo));
  }
  munmap(shared_data, sizeof(LinkTopoInfo));
  double detect_time = t1.Passed();

  LOG(INFO) << "DetectTopo Done, cost time: " << detect_time << "sec.";
}

std::vector<DistGraph::GroupConfig> PartitionSolver::solve() const  {
  IdType num_ctx = _ctxes.size();
  const auto &bandwidth_matrix = _topo_info.bandwidth_matrix;

  std::vector<std::vector<int>> access_count(
      num_ctx, std::vector<int>(num_ctx, 0));
  std::vector<std::vector<int>> access_part_ctx(
      num_ctx, std::vector<int>(num_ctx, -1));
  std::vector<std::set<int>> store_parts(num_ctx);

  std::vector<std::set<int>> can_access_parts(num_ctx);
  // from bandwidth matrix
  std::vector<std::set<int>> neighbor_adjacency(num_ctx);
  std::set<int> parts_universal_set;
  std::vector<std::tuple<int, int>> asc_degree_gpu_order(num_ctx);
  for (int i = 0; i < num_ctx; ++i) {
    parts_universal_set.insert(i);
    store_parts[i].insert(i);
    for (int j = 0; j < num_ctx; ++j) {
      if (bandwidth_matrix[i][j] != 0.0) {
        can_access_parts[i].insert(j);
        neighbor_adjacency[i].insert(j);
      }
    }
    asc_degree_gpu_order[i] = std::make_tuple(
        i, static_cast<int>(neighbor_adjacency[i].size()));
  }
  // sort nodes by ascending order to iterate
  std::sort(asc_degree_gpu_order.begin(), asc_degree_gpu_order.end(),
      [](auto x, auto y) {
        if (std::get<1>(x) != std::get<1>(y)) {
          return std::get<1>(x) < std::get<1>(y);
        }
        return std::get<0>(x) < std::get<0>(y);
      });
  std::stringstream ss;
  for (auto item : asc_degree_gpu_order) {
    ss << std::get<0>(item) << " ";
  }
  LOG(INFO) << "new node order to iterate: " << ss.str();
  // iterator for each GPU ctx
  for (auto item : asc_degree_gpu_order) {
    int i = std::get<0>(item);
    // get can not access parts for GPU i
    auto can_not_access_parts = (parts_universal_set - can_access_parts[i]);
    for (auto need_part : can_not_access_parts) {
      // id, stored_parts_size, need_score, if_same_part_in_neighbors, bandwidth
      std::vector<std::tuple<int, int, int, int, double>> tmp_vec;
      // iterate GPU_i neighbors
      for(auto j : neighbor_adjacency[i]) {
        int need_score = 0;
        for (auto k : neighbor_adjacency[j]) {
          if(!can_access_parts[k].count(need_part)) {
            ++need_score;
          }
        }
        tmp_vec.emplace_back(j, store_parts[j].size(), need_score,
            can_access_parts[j].count(need_part),
            bandwidth_matrix[i][j] / (access_count[i][j] + 1));
      }
      std::sort(tmp_vec.begin(), tmp_vec.end(), [](auto x, auto y){
            // stored_parts_size
            if (std::get<1>(x) != std::get<1>(y)) {
              return std::get<1>(x) < std::get<1>(y);
            }
            // need_score
            if (std::get<2>(x) != std::get<2>(y)) {
              return std::get<2>(x) > std::get<2>(y);
            }
            // if_same_part_in_neighbors 0 or 1
            if (std::get<3>(x) != std::get<3>(y)) {
              return std::get<3>(x) < std::get<3>(y);
            }
            // bandwidth
            if (std::get<4>(x) != std::get<4>(y)) {
              return std::get<4>(x) > std::get<4>(y);
            }
            return std::get<0>(x) < std::get<0>(y);
          });
      int choose_gpu_id = std::get<0>(tmp_vec.front());
      store_parts[choose_gpu_id].insert(need_part);
      // update can access parts for choose_gpu_id neighbors
      for (auto neighbor : neighbor_adjacency[choose_gpu_id]) {
        can_access_parts[neighbor].insert(need_part);
      }
    }
    // choose part in which GPU to access
    assert(can_access_parts[i].size() == num_ctx);
    for (int j = 0; j < num_ctx; ++j) {
      int which_gpu;
      double max_bandwidth = 0.0;
      for(auto neighbor : neighbor_adjacency[i]) {
        if (store_parts[neighbor].count(j)) {
          double tmp_bandwidth =
            bandwidth_matrix[i][neighbor] / (access_count[i][neighbor] + 1);
          if (tmp_bandwidth > max_bandwidth) {
            max_bandwidth = tmp_bandwidth;
            which_gpu = neighbor;
          }
        }
      }
      access_part_ctx[i][j] = which_gpu;
      access_count[i][which_gpu] += 1;
    }
  }

  std::vector<DistGraph::GroupConfig> configs;
  for (int i = 0; i < num_ctx; i++) {
    auto ctx = _ctxes[i];
    IdType device = ctx.device_id;
    CHECK_EQ(i, device);
    std::vector<IdType> part_ids(store_parts[device].begin(),
        store_parts[device].end());
    std::vector<Context> ctx_group(num_ctx);
    for (int j = 0; j < num_ctx; ++j) {
      ctx_group[j] = GPU(access_part_ctx[device][j]);
    }
    configs.emplace_back(ctx, part_ids, ctx_group);
  }
  return configs;
}

void PartitionSolver::DetectTopo_child(LinkTopoInfo *topo_info) {
  // 128M buffer for bandwidth test to detect backbone link
  size_t nbytes = (1<<27);
  IdType *buffers[kMaxDevice], *buffersD2D[kMaxDevice];
  hipStream_t stream[kMaxDevice];
  for (int i = 0; i < _ctxes.size(); i++) {
    int device = _ctxes[i].device_id;
    CUDA_CALL(hipSetDevice(device));
    // XXX: long time cost to lazy create ctx
    CUDA_CALL(hipMalloc(&buffers[device], nbytes));
    CUDA_CALL(hipMalloc(&buffersD2D[device], nbytes));
    CUDA_CALL(hipStreamCreateWithFlags(&stream[device], hipStreamNonBlocking));
    for (int j = 0; j < _ctxes.size(); j++) {
      int peer = _ctxes[j].device_id;
      topo_info->bandwidth_matrix[device][peer] = 0;
      if (device == peer) {
        topo_info->nvlink_matrix[device][peer] = 1;
        continue;
      }
      int can_access = false;
      CUDA_CALL(hipDeviceCanAccessPeer(&can_access, device, peer));
      if (!can_access) {
        topo_info->nvlink_matrix[device][peer] = 0;
      } else {
        topo_info->nvlink_matrix[device][peer] = 1;
      }
    }
  }
  for (int i = 0; i < _ctxes.size(); i++) {
    int device = _ctxes[i].device_id;
    CUDA_CALL(hipSetDevice(device));
    CUDA_CALL(hipMemcpyAsync(buffers[device], buffersD2D[device], nbytes, hipMemcpyDefault, stream[device]));
    CUDA_CALL(hipStreamSynchronize(stream[device]));
    for (int j = 0; j < _ctxes.size(); j++) {
      int peer = _ctxes[j].device_id;
      if (device != peer && topo_info->nvlink_matrix[device][peer]) {
        CUDA_CALL(hipDeviceEnablePeerAccess(peer, 0));
      }
    }
    for (int j = 0; j < _ctxes.size(); j++) {
      int peer = _ctxes[j].device_id;
      if (topo_info->nvlink_matrix[device][peer]) {
        Timer t0;
        CUDA_CALL(hipMemcpyAsync(buffers[device], buffersD2D[peer], nbytes, hipMemcpyDefault, stream[device]));
        CUDA_CALL(hipStreamSynchronize(stream[device]));
        auto sec = t0.Passed();
        if (device == peer) {
          topo_info->bandwidth_matrix[device][peer] = 2 * nbytes / sec / 1e9;
        } else {
          topo_info->bandwidth_matrix[device][peer] = nbytes / sec / 1e9;
        }
      }
    }
    for (int j = 0; j < _ctxes.size(); j++) {
      int peer = _ctxes[j].device_id;
      if (device != peer && topo_info->nvlink_matrix[device][peer]) {
        CUDA_CALL(hipDeviceDisablePeerAccess(peer));
      }
    }
  }

  // release resouce
  for (int i = 0; i < _ctxes.size(); i++) {
    auto device = _ctxes[i].device_id;
    CUDA_CALL(hipSetDevice(device));
    CUDA_CALL(hipStreamDestroy(stream[device]));
    CUDA_CALL(hipFree(buffers[device]));
    CUDA_CALL(hipFree(buffersD2D[device]));
  }

  std::stringstream ss;
  ss << "Topology Detect Debug: \n";
  for (int i = 0; i < _ctxes.size(); i++) {
    for (int j = 0; j < _ctxes.size(); j++) {
      ss << std::setw(8) << std::fixed << std::setprecision(1) << topo_info->bandwidth_matrix[i][j] << " ";
    }
    ss << "\n";
  }
  LOG(INFO) << ss.str();

  munmap(topo_info, sizeof(LinkTopoInfo));
  exit(0);
}

std::ostream& operator<<(std::ostream &os, const DistGraph::GroupConfig &config) {
  std::stringstream part_ss;
  std::stringstream peer_ss;
  for (auto part : config.part_ids)
    part_ss << part << " ";
  for (auto &ctx : config.ctx_group)
    peer_ss << ctx.device_id << " ";
  os << "GPU[" << config.ctx.device_id << "]"
     << " part: [ " << part_ss.str() << "]"
     << " peer: [ " << peer_ss.str() << "]";
  return os;
}

}  // namespace cuda
}  // namespace common
}  // namespace samgraph
