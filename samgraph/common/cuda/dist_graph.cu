#include "dist_graph.h"

#include <sys/mman.h>

#include <cstring>

#include "../device.h"

namespace samgraph {
namespace common {
namespace cuda {

std::shared_ptr<DistGraph> DistGraph::_inst = nullptr;

void DistGraph::_DatasetPartition(const Dataset *dataset, Context ctx,
    IdType part_id, IdType num_part) {
  auto indptr_data = dataset->indptr->CPtr<IdType>();
  auto indices_data = dataset->indices->CPtr<IdType>();
  IdType num_node = dataset->num_node;
  IdType part_edge_count = 0;
  for (IdType i = part_id; i < num_node; i += num_part) {
    IdType num_edge = (indptr_data[i + 1] - indptr_data[i]);
    part_edge_count += num_edge;
  }

  IdType indptr_size = (num_node / num_part +
      (part_id < num_node % num_part? 1 : 0) + 1);
  _part_indptr[part_id] = Tensor::Empty(kI32, {indptr_size}, CPU(),
      "indptr in device:" + std::to_string(ctx.device_id));
  _part_indices[part_id] = Tensor::Empty(kI32, {part_edge_count}, CPU(),
      "indices in device:" + std::to_string(ctx.device_id));
  part_edge_count = 0;

  for (IdType i = part_id; i < num_node; i += num_part) {
    IdType num_edge = (indptr_data[i + 1] - indptr_data[i]);
    IdType real_id = (i / num_part);
    _part_indptr[part_id]->Ptr<IdType>()[real_id] = part_edge_count;
    std::memcpy(
        &_part_indices[part_id]->Ptr<IdType>()[part_edge_count],
        &indices_data[indptr_data[i]],
        num_edge * sizeof(IdType));
    part_edge_count += num_edge;
  }
  _part_indptr[part_id]->Ptr<IdType>()[indptr_size - 1] = part_edge_count;

  _part_indptr[part_id] = Tensor::CopyTo(_part_indptr[part_id], ctx,
      nullptr, Constant::kAllocNoScale);
  _part_indices[part_id] = Tensor::CopyTo(_part_indices[part_id], ctx,
      nullptr, Constant::kAllocNoScale);
}

void DistGraph::DatasetLoad(Dataset *dataset, int sampler_id,
    Context sampler_ctx) {

  CHECK(sampler_ctx == _group_configs[sampler_id].ctx);
  _sampler_id = sampler_id;

  auto part_ids = _group_configs[sampler_id].part_ids;
  auto ctx_group = _group_configs[sampler_id].ctx_group;
  IdType num_part = ctx_group.size();
  _part_indptr.clear();
  _part_indptr.resize(num_part, nullptr);
  _part_indices.clear();
  _part_indices.resize(num_part, nullptr);

  for (IdType part_id : part_ids) {
    _DatasetPartition(dataset, sampler_ctx, part_id, num_part);
  }

  auto DataIpcShare = [&](std::vector<TensorPtr> &part_data,
      std::vector<size_t> part_size_vec,
      std::string name) {

    {
      for (IdType part_id : part_ids) {
        // share self data to others
        CHECK(sampler_ctx == part_data[part_id]->Ctx());
        CHECK(part_size_vec[part_id] == part_data[part_id]->Shape()[0]);
        auto shared_data = part_data[part_id]->CPtr<IdType>();
        hipIpcMemHandle_t &mem_handle =
          _shared_data->mem_handle[sampler_ctx.device_id][part_id];
        CUDA_CALL(hipIpcGetMemHandle(&mem_handle, (void*)shared_data));
      }
    }

    _Barrier();

    // receive data from others
    for (int i = 0; i < num_part; ++i) {
      if (part_data[i] != nullptr) {
        continue;
      }
      auto ctx = ctx_group[i];
      hipIpcMemHandle_t &mem_handle = _shared_data->mem_handle[ctx.device_id][i];
      void *ptr;
      CUDA_CALL(hipIpcOpenMemHandle(
            &ptr, mem_handle, hipIpcMemLazyEnablePeerAccess));
      part_data[i] = Tensor::FromBlob(ptr, kI32, {part_size_vec[i]}, ctx,
          name + " in device:" + std::to_string(ctx.device_id));
    }

  };

  IdType num_node = dataset->num_node;
  std::vector<size_t> part_size_vec(num_part);
  for (size_t i = 0; i < num_part; ++i) {
    part_size_vec[i] = (num_node / num_part +
        (i < num_node % num_part? 1 : 0) + 1);
  }
  DataIpcShare(_part_indptr, part_size_vec, "dataset part indptr");

  part_size_vec.clear();
  part_size_vec.resize(num_part, 0);
  auto indptr_data = dataset->indptr->CPtr<IdType>();
  for (IdType i = 0; i < num_node; ++i) {
    IdType num_edge = indptr_data[i + 1] - indptr_data[i];
    IdType tmp_part_id = (i % num_part);
    part_size_vec[tmp_part_id] += num_edge;
  }
  DataIpcShare(_part_indices, part_size_vec, "dataset part indices");

  CUDA_CALL(hipMalloc((void **)&_d_part_indptr, num_part * sizeof(IdType *)));
  CUDA_CALL(hipMalloc((void **)&_d_part_indices, num_part * sizeof(IdType *)));

  IdType **h_part_indptr, **h_part_indices;
  CUDA_CALL(hipHostMalloc(&h_part_indptr, num_part * sizeof(IdType*)));
  CUDA_CALL(hipHostMalloc(&h_part_indices, num_part * sizeof(IdType*)));
  for (IdType i = 0; i < num_part; i++) {
    h_part_indptr[i] = _part_indptr[i]->Ptr<IdType>();
    h_part_indices[i] = _part_indices[i]->Ptr<IdType>();
  }
  CUDA_CALL(hipMemcpy(_d_part_indptr, h_part_indptr, sizeof(IdType *) * num_part, hipMemcpyDefault));
  CUDA_CALL(hipMemcpy(_d_part_indices, h_part_indices, sizeof(IdType *) * num_part, hipMemcpyDefault));

  CUDA_CALL(hipHostFree(h_part_indptr));
  CUDA_CALL(hipHostFree(h_part_indices));

  _num_node = dataset->num_node;
}

DeviceDistGraph DistGraph::DeviceHandle() const {
  return DeviceDistGraph(
      _d_part_indptr, _d_part_indices,
      _group_configs[_sampler_id].ctx_group.size(),
      _num_node);
}

DistGraph::DistGraph(std::vector<Context> ctxes) {
  // TODO: from ctxes to get graph parts configs
  // bala bala ...
  std::vector<Context> ctx_group = ctxes;
  _group_configs.clear();
  for (int i = 0; i < ctxes.size(); ++i) {
    std::vector<int> part_ids = {i};
    _group_configs.emplace_back(ctxes[i], part_ids, ctx_group);
  }


  int num_worker = ctxes.size();
  _sampler_id = static_cast<int>(Constant::kEmptyKey);

  _shared_data = static_cast<SharedData*>(mmap(NULL, sizeof(SharedData),
                      PROT_READ|PROT_WRITE, MAP_SHARED|MAP_ANONYMOUS, -1, 0));
  CHECK_NE(_shared_data, MAP_FAILED);
  pthread_barrierattr_t attr;
  pthread_barrierattr_init(&attr);
  pthread_barrierattr_setpshared(&attr, PTHREAD_PROCESS_SHARED);
  pthread_barrier_init(&_shared_data->barrier, &attr, num_worker);
}

void DistGraph::_Barrier() {
  int err = pthread_barrier_wait(&_shared_data->barrier);
  CHECK(err == PTHREAD_BARRIER_SERIAL_THREAD || err == 0);
}

void DistGraph::Release(DistGraph *dist_graph) {
  if (dist_graph->_sampler_id != Constant::kEmptyKey) {
    for (int i = 0; i < dist_graph->_part_indptr.size(); i++) {
      if (i != dist_graph->_sampler_id) {
        CUDA_CALL(hipIpcCloseMemHandle(dist_graph->_part_indptr[i]->MutableData()));
        CUDA_CALL(hipIpcCloseMemHandle(dist_graph->_part_indices[i]->MutableData()));
      }
    }
    LOG(INFO) << "Release DistGraph" << " " << dist_graph->_sampler_id;
    // pthread_barrier_wait(&dist_graph->_shared_data->barrier);

    CUDA_CALL(hipFree((void*)dist_graph->_d_part_indptr));
    CUDA_CALL(hipFree((void*)dist_graph->_d_part_indices));
  }
  pthread_barrier_destroy(&dist_graph->_shared_data->barrier);
  munmap(dist_graph->_shared_data, sizeof(SharedData));
}

void DistGraph::Create(std::vector<Context> ctxes) {
  CHECK(_inst == nullptr);
  _inst = std::shared_ptr<DistGraph>(
      new DistGraph(ctxes), Release);
}

}  // namespace cuda
}  // namespace common
}  // namespace samgraph
