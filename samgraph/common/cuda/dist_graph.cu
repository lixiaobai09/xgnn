#include "dist_graph.h"

#include <sys/mman.h>
#include <sys/unistd.h>
#include <sys/wait.h>

#include <cstring>
#include <iomanip>
#include <set>
#include <algorithm>

#include "../device.h"
#include "../timer.h"


namespace samgraph {
namespace common {
namespace cuda {

std::shared_ptr<DistGraph> DistGraph::_inst = nullptr;

void DistGraph::_DatasetPartition(const Dataset *dataset, int sampler_id) {
  auto indptr_data = dataset->indptr->CPtr<IdType>();
  auto indices_data = dataset->indices->CPtr<IdType>();
  auto ctx = _group_configs[sampler_id].ctx;
  IdType num_node = dataset->num_node;
  IdType num_part = _group_configs[sampler_id].ctx_group.size();
  IdType part_id = _group_configs[sampler_id].part_id;
  IdType part_edge_count = 0;
  for (IdType i = part_id; i < num_node; i += num_part) {
    IdType num_edge = (indptr_data[i + 1] - indptr_data[i]);
    part_edge_count += num_edge;
  }
  _part_indptr.clear();
  _part_indptr.resize(num_part, nullptr);
  _part_indices.clear();
  _part_indices.resize(num_part, nullptr);

  IdType indptr_size = (num_node / num_part +
      (part_id < num_node % num_part? 1 : 0) + 1);
  _part_indptr[part_id] = Tensor::Empty(kI32, {indptr_size}, CPU(),
      "indptr in device:" + std::to_string(ctx.device_id));
  _part_indices[part_id] = Tensor::Empty(kI32, {part_edge_count}, CPU(),
      "indices in device:" + std::to_string(ctx.device_id));
  part_edge_count = 0;

  for (IdType i = part_id; i < num_node; i += num_part) {
    IdType num_edge = (indptr_data[i + 1] - indptr_data[i]);
    IdType real_id = (i / num_part);
    _part_indptr[part_id]->Ptr<IdType>()[real_id] = part_edge_count;
    std::memcpy(
        &_part_indices[part_id]->Ptr<IdType>()[part_edge_count],
        &indices_data[indptr_data[i]],
        num_edge * sizeof(IdType));
    part_edge_count += num_edge;
  }
  _part_indptr[part_id]->Ptr<IdType>()[indptr_size - 1] = part_edge_count;

  _part_indptr[part_id] = Tensor::CopyTo(_part_indptr[part_id], ctx,
      nullptr, Constant::kAllocNoScale);
  _part_indices[part_id] = Tensor::CopyTo(_part_indices[part_id], ctx,
      nullptr, Constant::kAllocNoScale);
}

void DistGraph::DatasetLoad(Dataset *dataset, int sampler_id,
    Context sampler_ctx) {

  CHECK(sampler_ctx == _group_configs[sampler_id].ctx);

  _sampler_id = sampler_id;
  _DatasetPartition(dataset, sampler_id);

  auto ctx_group = _group_configs[sampler_id].ctx_group;
  IdType part_id = _group_configs[sampler_id].part_id;
  IdType num_part = ctx_group.size();

  auto DataIpcShare = [&](std::vector<TensorPtr> &part_data,
      std::vector<size_t> part_size_vec,
      std::string name) {

    {
      // share self data to others
      CHECK(sampler_ctx == part_data[part_id]->Ctx());
      auto shared_data = part_data[part_id]->CPtr<IdType>();
      hipIpcMemHandle_t &mem_handle =
        _shared_data->mem_handle[sampler_ctx.device_id];
      CUDA_CALL(hipIpcGetMemHandle(&mem_handle, (void*)shared_data));
    }

    _Barrier();

    // receive data from others
    for (int i = 0; i < num_part; ++i) {
      if (i == part_id) {
        continue;
      }
      CHECK(part_data[i] == nullptr);
      auto ctx = ctx_group[i];
      hipIpcMemHandle_t &mem_handle = _shared_data->mem_handle[ctx.device_id];
      void *ptr;
      CUDA_CALL(hipIpcOpenMemHandle(
            &ptr, mem_handle, hipIpcMemLazyEnablePeerAccess));
      part_data[i] = Tensor::FromBlob(ptr, kI32, {part_size_vec[i]}, ctx,
          name + " in device:" + std::to_string(ctx.device_id));
    }

  };

  IdType num_node = dataset->num_node;
  std::vector<size_t> part_size_vec(num_part);
  for (size_t i = 0; i < num_part; ++i) {
    part_size_vec[i] = (num_node / num_part +
        (i < num_node % num_part? 1 : 0) + 1);
  }
  DataIpcShare(_part_indptr, part_size_vec, "dataset part indptr");

  part_size_vec.clear();
  part_size_vec.resize(num_part, 0);
  auto indptr_data = dataset->indptr->CPtr<IdType>();
  for (IdType i = 0; i < num_node; ++i) {
    IdType num_edge = indptr_data[i + 1] - indptr_data[i];
    IdType part_id = (i % num_part);
    part_size_vec[part_id] += num_edge;
  }
  DataIpcShare(_part_indices, part_size_vec, "dataset part indices");

  CUDA_CALL(hipMalloc((void **)&_d_part_indptr, num_part * sizeof(IdType *)));
  CUDA_CALL(hipMalloc((void **)&_d_part_indices, num_part * sizeof(IdType *)));

  IdType **h_part_indptr, **h_part_indices;
  CUDA_CALL(hipHostMalloc(&h_part_indptr, num_part * sizeof(IdType*)));
  CUDA_CALL(hipHostMalloc(&h_part_indices, num_part * sizeof(IdType*)));
  for (IdType i = 0; i < num_part; i++) {
    h_part_indptr[i] = _part_indptr[i]->Ptr<IdType>();
    h_part_indices[i] = _part_indices[i]->Ptr<IdType>();
  }
  CUDA_CALL(hipMemcpy(_d_part_indptr, h_part_indptr, sizeof(IdType *) * num_part, hipMemcpyDefault));
  CUDA_CALL(hipMemcpy(_d_part_indices, h_part_indices, sizeof(IdType *) * num_part, hipMemcpyDefault));

  CUDA_CALL(hipHostFree(h_part_indptr));
  CUDA_CALL(hipHostFree(h_part_indices));

  _num_node = dataset->num_node;
}

DeviceDistGraph DistGraph::DeviceHandle() const {
  return DeviceDistGraph(
      _d_part_indptr, _d_part_indices,
      _group_configs[_sampler_id].ctx_group.size(),
      _num_node);
}

DistGraph::DistGraph(std::vector<Context> ctxes) {
  // TODO: from ctxes to get graph parts configs
  // bala bala ...
  std::vector<Context> ctx_group = ctxes;

  PartitionSolver solver(ctx_group);
  auto configs = solver.solve();
  for (auto &config : configs) {
    LOG(INFO) << config;
  }

  _group_configs.clear();
  for (int i = 0; i < ctxes.size(); ++i) {
    _group_configs.emplace_back(ctxes[i], i, ctx_group);
  }


  int num_worker = ctxes.size();
  _sampler_id = static_cast<int>(Constant::kEmptyKey);

  _shared_data = static_cast<SharedData*>(mmap(NULL, sizeof(SharedData),
                      PROT_READ|PROT_WRITE, MAP_SHARED|MAP_ANONYMOUS, -1, 0));
  CHECK_NE(_shared_data, MAP_FAILED);
  pthread_barrierattr_t attr;
  pthread_barrierattr_init(&attr);
  pthread_barrierattr_setpshared(&attr, PTHREAD_PROCESS_SHARED);
  pthread_barrier_init(&_shared_data->barrier, &attr, num_worker);
}

void DistGraph::_Barrier() {
  int err = pthread_barrier_wait(&_shared_data->barrier);
  CHECK(err == PTHREAD_BARRIER_SERIAL_THREAD || err == 0);
}

void DistGraph::Release(DistGraph *dist_graph) {
  if (dist_graph->_sampler_id != Constant::kEmptyKey) {
    for (int i = 0; i < dist_graph->_part_indptr.size(); i++) {
      if (i != dist_graph->_sampler_id) {
        CUDA_CALL(hipIpcCloseMemHandle(dist_graph->_part_indptr[i]->MutableData()));
        CUDA_CALL(hipIpcCloseMemHandle(dist_graph->_part_indices[i]->MutableData()));
      }
    }
    LOG(INFO) << "Release DistGraph" << " " << dist_graph->_sampler_id;
    // pthread_barrier_wait(&dist_graph->_shared_data->barrier);

    CUDA_CALL(hipFree((void*)dist_graph->_d_part_indptr));
    CUDA_CALL(hipFree((void*)dist_graph->_d_part_indices));
  }
  pthread_barrier_destroy(&dist_graph->_shared_data->barrier);
  munmap(dist_graph->_shared_data, sizeof(SharedData));
}

void DistGraph::Create(std::vector<Context> ctxes) {
  CHECK(_inst == nullptr);
  _inst = std::shared_ptr<DistGraph>(
      new DistGraph(ctxes), Release);
}


PartitionSolver::PartitionSolver(const std::vector<Context> &ctx_group) 
  : _ctx_group(ctx_group) {
  std::set<int> set;
  for (auto&ctx : ctx_group) {
    set.insert(ctx.device_id);
  }
  CHECK_EQ(set.size(), ctx_group.size());
  CHECK_EQ(*set.rbegin() + 1, set.size());
  DetectTopo();
}

void PartitionSolver::DetectTopo() {
  // shared memory for transfer detect result
  LinkTopoInfo *shared_data = (LinkTopoInfo*)mmap(NULL, sizeof(LinkTopoInfo), 
    PROT_WRITE | PROT_READ, MAP_SHARED | MAP_ANONYMOUS, -1, 0);
  int pid = fork();
  CHECK(pid != -1);
  if (pid == 0) {
    DetectTopo_child(shared_data);
  } else {
    int wstatus;
    waitpid(pid, &wstatus, 0);
    if (WEXITSTATUS(wstatus) != 0 || WIFSIGNALED(wstatus)) {
      CHECK(false);
    }
    std::memcpy(&_topo_info, shared_data, sizeof(LinkTopoInfo));
  }
  munmap(shared_data, sizeof(LinkTopoInfo));

  LOG(INFO) << "DetectTopo Done";
}

std::vector<PartitionSolver::GroupConfig> PartitionSolver::solve() const  {
  std::set<IdType> parts[kMaxDevice];
  IdType access_matrix[kMaxDevice][kMaxDevice] = {0};
  IdType access_cnt[kMaxDevice][kMaxDevice] = {0};
  
  for (int i = 0; i < _ctx_group.size(); i++) {
    int device = _ctx_group[i].device_id;
    parts[device].insert(device);
  }
  
  auto neighborParts = [&](IdType device, IdType part) -> std::vector<IdType> {
    std::vector<IdType> peer_vec;
    for (int peer = 0; peer < this->_ctx_group.size(); peer++) {
      if (this->_topo_info.nvlink_matrix[device][peer]) {
        if (parts[peer].find(part) != parts[peer].end()) {
          peer_vec.push_back(peer);
        } 
      }
    }
    return peer_vec;
  };

  for (int device = 0; device < _ctx_group.size(); device++) {
    std::vector<IdType> miss_parts;
    for (int part = 0; part < _ctx_group.size(); part++) {
      auto peers = neighborParts(device, part);
      if (peers.empty()) {
        miss_parts.push_back(part);
      } else {
        auto peer = ChoosePeer(parts, access_cnt, device, peers, true);
        access_cnt[device][peer]++;
        access_matrix[device][part] = peer;
      }
    }
    for (auto part : miss_parts) {
      IdType rep_peer = FindPalcement(parts, access_cnt, device, part);
      parts[rep_peer].insert(part);
      access_cnt[device][rep_peer]++;
      access_matrix[device][part] = rep_peer;
    }
  }
  std::vector<PartitionSolver::GroupConfig> configs;
  for (int i = 0; i < _ctx_group.size(); i++) {
    auto ctx = _ctx_group[i];
    IdType device = ctx.device_id;
    std::vector<IdType> part_ids(parts[device].begin(), parts[device].end());
    std::vector<Context> group;
    for (int j = 0; j < _ctx_group.size(); j++) {
      group.push_back(GPU(access_matrix[device][j]));
    }
    configs.emplace_back(ctx, part_ids, group);
  }
  return configs;
}

IdType PartitionSolver::FindPalcement(
  const std::set<IdType> parts[], IdType access_cnt[][kMaxDevice],
  IdType device, IdType part) const {
  std::vector<IdType> peers;
  for (IdType peer = 0; peer < _ctx_group.size(); peer++) {
    if (_topo_info.nvlink_matrix[device][peer]) {
      peers.push_back(peer);
    }
  }
  CHECK(peers.size() > 0);
  return ChoosePeer(parts, access_cnt, device, peers, false);
}

IdType PartitionSolver::ChoosePeer(
  const std::set<IdType> parts[], IdType access_cnt[][kMaxDevice],
  IdType device, std::vector<IdType> peers, bool exist) const {
  if (peers.empty()) {
    return -1;
  }
  std::vector<std::pair<IdType, double>> weight;
  for (auto peer : peers) {
    double bw = _topo_info.bandwitdh_matrix[device][peer];
    bw /= (access_cnt[device][peer] + exist);
    weight.push_back({parts[peer].size(), bw});
  }
  std::sort(peers.begin(), peers.end(), [&](IdType x, IdType y) {
    if (weight[x].first != weight[y].first) {
      return weight[x].first < weight[y].first;
    } else {
      return weight[x].second > weight[y].second;
    }
  });
  return peers.front();
}

void PartitionSolver::DetectTopo_child(LinkTopoInfo *topo_info) {
  // 128M buffer for bandwidth test to detect backbone link
  size_t nbytes = (1<<27);
  IdType *buffers[kMaxDevice], *buffersD2D[kMaxDevice];
  hipStream_t stream[kMaxDevice];
  for (int i = 0; i < _ctx_group.size(); i++) {
    int device = _ctx_group[i].device_id;
    CUDA_CALL(hipSetDevice(device));
    CUDA_CALL(hipMalloc(&buffers[device], nbytes));
    CUDA_CALL(hipMalloc(&buffersD2D[device], nbytes));
    CUDA_CALL(hipStreamCreateWithFlags(&stream[device], hipStreamNonBlocking));
    for (int j = 0; j < _ctx_group.size(); j++) {
      int peer = _ctx_group[j].device_id;
      topo_info->bandwitdh_matrix[device][peer] = 0;
      if (device == peer) {
        topo_info->nvlink_matrix[device][peer] = 1;
        continue;
      }
      int can_access = false;
      CUDA_CALL(hipDeviceCanAccessPeer(&can_access, device, peer));
      if (!can_access) {
        topo_info->nvlink_matrix[device][peer] = 0;
      } else {
        topo_info->nvlink_matrix[device][peer] = 1;
      }
    }
  }
  for (int i = 0; i < _ctx_group.size(); i++) {
    int device = _ctx_group[i].device_id;
    CUDA_CALL(hipSetDevice(device));
    CUDA_CALL(hipMemcpyAsync(buffers[device], buffersD2D[device], nbytes, hipMemcpyDefault, stream[device]));
    CUDA_CALL(hipStreamSynchronize(stream[device]));
    for (int j = 0; j < _ctx_group.size(); j++) {
      int peer = _ctx_group[j].device_id;
      if (device != peer && topo_info->nvlink_matrix[device][peer]) {
        CUDA_CALL(hipDeviceEnablePeerAccess(peer, 0));
      }
    }
    for (int j = 0; j < _ctx_group.size(); j++) {
      int peer = _ctx_group[j].device_id;
      if (topo_info->nvlink_matrix[device][peer]) {
        Timer t0;
        CUDA_CALL(hipMemcpyAsync(buffers[device], buffersD2D[peer], nbytes, hipMemcpyDefault, stream[device]));
        CUDA_CALL(hipStreamSynchronize(stream[device]));
        auto sec = t0.Passed();
        if (device == peer) {
          topo_info->bandwitdh_matrix[device][peer] = 2 * nbytes / sec / 1e9;
        } else {
          topo_info->bandwitdh_matrix[device][peer] = nbytes / sec / 1e9;
        }
      }
    }
    for (int j = 0; j < _ctx_group.size(); j++) {
      int peer = _ctx_group[j].device_id;
      if (device != peer && topo_info->nvlink_matrix[device][peer]) {
        CUDA_CALL(hipDeviceDisablePeerAccess(peer));
      }
    }
  }

  // release resouce
  for (int i = 0; i < _ctx_group.size(); i++) {
    auto device = _ctx_group[i].device_id;
    CUDA_CALL(hipSetDevice(device));
    CUDA_CALL(hipStreamDestroy(stream[device]));
    CUDA_CALL(hipFree(buffers[device]));
    CUDA_CALL(hipFree(buffersD2D[device]));
    for (int j = 0; j < _ctx_group.size(); j++) {
      auto peer = _ctx_group[j].device_id;
      if (device == peer)
        continue;
    }
  }

  std::stringstream ss;
  ss << "Topology Detect Debug: \n";
  for (int i = 0; i < _ctx_group.size(); i++) {
    for (int j = 0; j < _ctx_group.size(); j++) {
      ss << std::setw(8) << std::fixed << std::setprecision(1) << topo_info->bandwitdh_matrix[i][j] << " ";
    }
    ss << "\n";
  }
  LOG(INFO) << ss.str();

  munmap(topo_info, sizeof(LinkTopoInfo));
  exit(0);
}

std::ostream& operator<<(std::ostream &os, const PartitionSolver::GroupConfig &config) {
  std::stringstream part_ss;
  std::stringstream peer_ss;
  for (auto part : config.part_ids)
    part_ss << part << " ";
  for (auto &ctx : config.ctx_group)
    peer_ss << ctx.device_id << " ";
  os << "GPU[" << config.ctx.device_id << "]"
     << " part: [ " << part_ss.str() << "]"
     << " peer: [ " << peer_ss.str() << "]";
  return os;
}

}  // namespace cuda
}  // namespace common
}  // namespace samgraph
