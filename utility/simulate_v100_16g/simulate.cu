
#include <hip/hip_runtime.h>
#include <iostream>
#include <unistd.h>
using namespace std;

#define CUDA_CALL(func)                         \
 {                                              \
    hipError_t err = func;                     \
    if(err != hipSuccess) {                    \
        cout << __FILE__ << ":" << __LINE__     \
             << " " << #func << " "             \
             << hipGetErrorString(err)         \
             << " errnum " << err;              \
        exit(EXIT_FAILURE);                     \
    }                                           \
 }

int main() {
    int *arr[8];
    for (int i = 0; i < 8; i++) {
        CUDA_CALL(hipSetDevice(i));
        CUDA_CALL(hipMalloc(&arr[i], 16859004928));
    }

    while(1) {
        sleep(10);
    }
}